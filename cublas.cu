#include "hip/hip_runtime.h"
/*!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
! This file is part of nsCouette -- A high-performance code for direct         !
! numerical simulations of turbulent Taylor-Couette flow                       !
!                                                                              !
! Copyright (C) 2019 Marc Avila, Bjoern Hof, Jose Manuel Lopez, Markus Rampp,  !
!                    Liang Shi, Alberto Vela-Martin, Daniel Feldmann.          !
!                                                                              !
! nsCouette is free software: you can redistribute it and/or modify it under   !
! the terms of the GNU General Public License as published by the Free         !
! Software Foundation, either version 3 of the License, or (at your option)    !
! any later version.                                                           !
!                                                                              !
! nsCouette is distributed in the hope that it will be useful, but WITHOUT ANY !
! WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS    !
! FOR A PARTICULAR PURPOSE. See the GNU General Public License for more        !
! details.                                                                     !
!                                                                              !
! You should have received a copy of the GNU General Public License along with !
! nsCouette. If not, see <http://www.gnu.org/licenses/>.                       !
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!*/

#include"TayC.h"

static hipblasHandle_t cublasHandle;
static double2 alpha[1];
static double alphad[1];
static int* infoArray;
static double *dirM, *invM;
double *Ah[NT*NZ];
double *Bh[NT*NZ];
double** dirMp,**invMp;
int* pivotArray;

void cublasCheck(hipblasStatus_t error, const char* function )
{
  if(error !=  HIPBLAS_STATUS_SUCCESS)
  {
    printf("\n error  %s : %d \n", function, error);
    exit(1);
  }
    
  return;
}  

void setCublas(void){

    cublasCheck(hipblasCreate(&cublasHandle),"Cre");


    CHECK_CUDART(hipMalloc((void**)&infoArray,NT*NZ*sizeof(int)));

    alpha[0].x=1.0;
    alpha[0].y=0.0;

    alphad[0]=1.0;

    //Set to invert infinity matrix
        
    CHECK_CUDART(hipMalloc((void **)&dirM,8*8*NZ*NT*sizeof(double)));
    CHECK_CUDART(hipMalloc((void **)&invM,8*8*NZ*NT*sizeof(double)));
    
    CHECK_CUDART(hipMalloc((void **)&dirMp,NZ*NT*sizeof(double*)));
    CHECK_CUDART(hipMalloc((void **)&invMp,NZ*NT*sizeof(double*)));
    
    CHECK_CUDART(hipMalloc((void **)&pivotArray,NZ*NT*sizeof(int)));

    
    for(int i=0;i<NT*NZ;i++){
        Ah[i] = &dirM[i*8*8];
        Bh[i] = &invM[i*8*8];
    }

    CHECK_CUDART(hipMemcpy(dirMp,Ah,NT*NZ*sizeof(double*),hipMemcpyHostToDevice));
    CHECK_CUDART(hipMemcpy(invMp,Bh,NT*NZ*sizeof(double*),hipMemcpyHostToDevice));
    
    return;
    }

void transpose_A(double2* u_2,double2* u_1){

	//Transpuesta de [i,k,j][NZ,NT,NR] a -----> [j,i,k][NR,NZ,NT]

	cublasCheck(hipblasZgeam(cublasHandle,HIPBLAS_OP_T,HIPBLAS_OP_T,NZ*NT,NR,
                            alpha,(const double2*)u_1,NR,0,0,NR,u_2,NZ*NT),"Tr");
	return;


}

void transpose_infforward(double* u_2, double* u_1,int mr){

	//Transpuesta de [j,i,k][mr,NZ,NT] a -----> [i,k,j][NZ,NT,mr]

	cublasCheck(hipblasDgeam(cublasHandle,HIPBLAS_OP_T,HIPBLAS_OP_T,mr,NZ*NT,
                            alphad,(const double*)u_1,NZ*NT,0,0,NZ*NT,u_2,mr),"Tr");
	return;

}

void transpose_infbackward(double* u_2,double* u_1,int mr){

	//Transpuesta de [i,k,j][NZ,NT,mr] a -----> [j,i,k][mr,NZ,NT]

	cublasCheck(hipblasDgeam(cublasHandle,HIPBLAS_OP_T,HIPBLAS_OP_T,NZ*NT,mr,
                            alphad,(const double*)u_1,mr,0,0,mr,u_2,NZ*NT),"Tr");
	return;


}

void invert_infmatrix(double* src){


    transpose_infforward(dirM,src,8*8);
    
    cublasCheck(cublasDmatinvBatched(cublasHandle,8,(const double**)dirMp,8,
                   (double**)invMp,8,infoArray,NT*NZ),"Tr");
     
    transpose_infbackward(src,invM,8*8);


     return;
}
    
