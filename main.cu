#include "hip/hip_runtime.h"
/*!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
! This file is part of nsCouette -- A high-performance code for direct         !
! numerical simulations of turbulent Taylor-Couette flow                       !
!                                                                              !
! Copyright (C) 2019 Marc Avila, Bjoern Hof, Jose Manuel Lopez, Markus Rampp,  !
!                    Liang Shi, Alberto Vela-Martin, Daniel Feldmann.          !
!                                                                              !
! nsCouette is free software: you can redistribute it and/or modify it under   !
! the terms of the GNU General Public License as published by the Free         !
! Software Foundation, either version 3 of the License, or (at your option)    !
! any later version.                                                           !
!                                                                              !
! nsCouette is distributed in the hope that it will be useful, but WITHOUT ANY !
! WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS    !
! FOR A PARTICULAR PURPOSE. See the GNU General Public License for more        !
! details.                                                                     !
!                                                                              !
! You should have received a copy of the GNU General Public License along with !
! nsCouette. If not, see <http://www.gnu.org/licenses/>.                       !
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!*/

#include"TayC.h"

int main( int argc, const char* argv[]){
  
  printf("\n+++++++++++++++++++++++++++");
  printf("\nStarting GPU Taylor-Couette");
  printf("\n+++++++++++++++++++++++++++");
  
  int dev=7;
  double time;

  printf("\n");
  printf("\nSetting device: %d",dev);
  CHECK_CUDART(hipSetDevice(dev));
  
  //Timing variables
  clock_t start, end;
  double gpu_time_used;

  //Set up
  size_p sizes;
    
  sizes.Nr=NR;
  sizes.Nt=NT;
  sizes.Nz=NZ;
    
  //Initialize the mesh  
  double* grid_mesh=(double*)malloc(NR*sizeof(double));

  double r_i = 1.0;         //ETA/(1.0-ETA);
  double r_o = 1.0/ETA;     //1.0/(1.0-ETA);
  
  
  //Chebyshev mesh in r
  
  for(int j=0;j<NR;j++){
      //grid_mesh[j]=2.0*(double)j/(double)(NR-1)+1.0;
        grid_mesh[j]= (r_i+r_o)/2.0 - cos(PI2/2.0*j/(NR-1))/2.0;
    
  }
  
  //Write mesh

  //Set modules
  setImplic_7_exp(grid_mesh,sizes);
  setDeriv_9_exp(grid_mesh,sizes);
  setBoundary(grid_mesh);
  setNonlinear(grid_mesh);
  setFft(sizes);
  setCublas();
  setIntegrator(sizes);
  setLinear(grid_mesh);
  setStatistics(grid_mesh);
  
  //Allocate memory buffers
  vfield u, uw, rhs, rhsw;
  
  size_t size_p=NR*NT*NZ*sizeof(double2);

  CHECK_CUDART(hipMalloc(&u.r,size_p));
  CHECK_CUDART(hipMalloc(&u.t,size_p));
  CHECK_CUDART(hipMalloc(&u.z,size_p));
  
  CHECK_CUDART(hipMalloc(&uw.r,size_p));
  CHECK_CUDART(hipMalloc(&uw.t,size_p));
  CHECK_CUDART(hipMalloc(&uw.z,size_p));
  
  CHECK_CUDART(hipMalloc(&rhs.r,size_p));
  CHECK_CUDART(hipMalloc(&rhs.t,size_p));
  CHECK_CUDART(hipMalloc(&rhs.z,size_p));
  
  CHECK_CUDART(hipMalloc(&rhsw.r,size_p));
  CHECK_CUDART(hipMalloc(&rhsw.t,size_p));
  CHECK_CUDART(hipMalloc(&rhsw.z,size_p));

  
  //Start initial field
  initField(u, grid_mesh);
  
  //Or read check point
  //readCheckpoint(u,grid_mesh,&time,"./checkpoint.h5");

  //Time-step
  printf("\nParameter");
  printf("\nr_o,r_i=%e,%e",r_o,r_i);
  printf("\nRe_i,Re_o=%e,%e",REYNOLDS_OUTER,REYNOLDS_INNER);
  
  double U_i=REYNOLDS_OUTER*r_o;
  double U_o=REYNOLDS_INNER*r_i;
  
  double t_i=PI2*r_i/NT/U_i;
  double t_o=PI2*r_o/NT/U_o;
 
  printf("\nt_i,t_o=%e,%e",t_o,t_i);
  
  double dt=0.1*min(abs(t_i),abs(t_o));
  
  if(VARIABLE_DT){
    printf("\nRunning with fixed Courant number=%e",COURANT);
  }else{
    printf("\nRunning with fixed Dt=%e",dt);
  }
  
  //Number of steps
  int nsteps=200;
  
  start=clock(); 

  integrate(u, uw, rhs, rhsw, nsteps, dt,&time);
  
  CHECK_CUDART(hipDeviceSynchronize());
  end=clock();  

  time=dt*nsteps;

  printf("\nTime_per_step=%e s",((double)(end-start))/CLOCKS_PER_SEC/nsteps);  

  //writeCheckpoint(u,grid_mesh,&time,"./checkpoint.h5");
  //writeFieldVis(u,grid_mesh,&time,"./visualiza.h5");


  return 0;
  
}
