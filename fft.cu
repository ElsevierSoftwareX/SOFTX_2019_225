#include "hip/hip_runtime.h"
#include "TayC.h"

/*static hipfftHandle fft2_r2c; 
static hipfftHandle fft2_c2r;*/ 

static hipfftHandle fft2_r2c_zp; 
static hipfftHandle fft2_c2r_zp;

static hipfftHandle fft2_c2r;
static hipfftHandle fft2_r2c;

static double2* aux_fft;

static size_t sizep;

void cufftCheck( hipfftResult error, const char* function )
{
	if(error != HIPFFT_SUCCESS)
	{
		printf("\n error  %s : %d \n", function, error);
		exit(1);
	}
		
	return;
}  

void setFft(size_p sizes)
{

    sizep = sizeof(double2)*NR*NZP*NTP;

    int nzp[2]={NTP,2*NZP-2};

    cufftCheck(hipfftPlanMany( &fft2_r2c_zp,2,nzp,NULL,1,0,NULL,1,0,HIPFFT_D2Z,NR/NSTEPS_CONV),"ALLOCATE_FFT3_R2C_ZP");
    cufftCheck(hipfftPlanMany( &fft2_c2r_zp,2,nzp,NULL,1,0,NULL,1,0,HIPFFT_Z2D,NR/NSTEPS_CONV),"ALLOCATE_FFT3_C2R_ZP");

    int nz[2]={NT,2*NZ-2};
    
    cufftCheck(hipfftPlanMany( &fft2_c2r,2,nz,NULL,1,0,NULL,1,0,HIPFFT_Z2D,NR),"ALLOCATE_FFT3_R2C_ZP");
    cufftCheck(hipfftPlanMany( &fft2_r2c,2,nz,NULL,1,0,NULL,1,0,HIPFFT_D2Z,NR),"ALLOCATE_FFT3_C2R_ZP");

    
    if(NR%NSTEPS_CONV!=0){
        printf("\nError tama?os:NR must be divisible by NSTEPS_CONV");exit(1);
    }

    CHECK_CUDART(hipMalloc((void**)&aux_fft,sizep));
      
    return;
}

void fftDestroy(void)
{
  hipfftDestroy(fft2_r2c_zp);
  hipfftDestroy(fft2_c2r_zp);

  return;
}

void fftForward(double2* buffer)
{
//   cufftCheck(hipfftExecD2Z(fft2_r2c_zp,(double*)buffer,(double2*)aux_fft),"forward transform_zp");
//   normalize(aux_fft,(double)NTP*(2*NZP-2));
//   CHECK_CUDART(hipMemcpy(buffer,aux_fft,sizep,hipMemcpyDeviceToDevice));

     cufftCheck(hipfftExecD2Z(fft2_r2c_zp,(double*)buffer,(double2*)buffer),"backward transform_zp");
     normalize(buffer,(double)NTP*(2*NZP-2),NR*NTP*NZP);

    //Normalize
    
  return;
}

void fftBackward(double2* buffer)
{
//   cufftCheck(hipfftExecZ2D(fft2_c2r_zp,(double2*)buffer,(double*)aux_fft),"backward transform_zp");
//   CHECK_CUDART(hipMemcpy(buffer,aux_fft,sizep,hipMemcpyDeviceToDevice));

  cufftCheck(hipfftExecZ2D(fft2_c2r_zp,(double2*)buffer,(double*)buffer),"backward transform_zp");
  //normalize(buffer,(double)NTP*(2*NZP-2),NR*NTP*NZP);

  //   CHECK_CUDART(hipMemcpy(buffer,aux_fft,sizep,hipMemcpyDeviceToDevice));
  
  return;

}

void fftBackward_reduced(double2* buffer)
{
  
  cufftCheck(hipfftExecZ2D(fft2_c2r,(double2*)buffer,(double*)buffer),"backward transform");
//   normalize(buffer,(double)NT*(2*NZ-2),NR*NT*NZ);
  
  return;

}

void fftForward_reduced(double2* buffer)
{
  
  cufftCheck(hipfftExecD2Z(fft2_r2c,(double*)buffer,(double2*)buffer),"backward transform");
  normalize(buffer,(double)NT*(2*NZ-2),NR*NT*NZ);

  return;

}
